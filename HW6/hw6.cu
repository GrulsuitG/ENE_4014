#include "hip/hip_runtime.h"
/*
 * Find BLANK and replace your own code.
 * And submit report why do you replace the blank that way.
 */

#include<stdlib.h>
#include<iostream>
#include<fstream>
#include<vector>
#include<string>

#define TILE_WIDTH_TIMES 1 // fix this to evalute computation speed
#define INPUT_FILENAME "input.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpolling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // out of bound

    

    int bound = input_size / filter_size ;
    int max = 0;
    
    int global_x = 0 ;
    int global_y = 0;

    for(int i = 0; i < filter_size; i++){
        if( i * bound <= col && col < (i+1) * bound)
            global_x = i;
        if( i * bound <= row && row < (i+1) * bound)
            global_y = i;
    }
    cout << "(" << row << "," << row << ")" << " = " << "(" << global_x << "," << global_y << ")" << "\n";
    
}

__global__ void avgpool(float *input, float *output, const int input_size, const int filter_size) {
    // input : input_matrix address
    // output : output buffer address
    // input_size : width, height of input matrix
    // filter_size : filter_size of maxpolling
    // all input, output matrices are vectorized

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // allocate 2D tiles in __shared__ memory
    // TILE WIDTH = TILE_WIDTH_TIMES*filter_size
    int tile_width = TILE_WIDTH_TIMES * filter_size;

    __shared__ float sum_tile[tile_width][tile_width];

    float result = 0;

    // make sure you handle the case when the matrix sizes are not
    // multiple of tile_width!
    // loop over the tiles of the input in phases
    for(int p = 0; p < input_size/tile_width; ++p){
        // CHANGE

        // You need to use __syncthreads() a few times
        // to synchronize the threads in a thread block.
    }

    // write out the result to output[row*input_size + col] 
    // CHANGE
}

int main(int argc, char **argv) {
    if(argc < 2) {
        cout << "usage : " << argv[0] << " input_size filter_size\n" << "example : " << argv[0] << " 100 2\n";
        return 1;
    }
    const int input_size = stoi(argv[1]);
    const int filter_size = stoi(argv[2]); // used for maxpooling
    
    const int maxpool_output_size = input_size/filter_size;
    const int avgpool_output_size = input_size/filter_size;

    // check input_size is power of 2
    if(input_size == 0 && (input_size & (input_size-1))){
        cout << "input_size must be power of 2\n";
        return 1;
    }

    if(filter_size == 0){
        cout << "filter_size cannot be 0\n";
        return 1;
    }

    float pool_input[input_size*input_size];
    
    // read input matrices 
    ifstream input_in(INPUT_FILENAME);

    for (int i = 0; i < input_size*input_size; ++i) {
        input_in >> pool_input[i];
    }
    
    // prints inputs for debugging.
    cout<<"filter size : "<<filter_size;
    cout<<"\n========== POOL_INPUT ==========\n";
    for (int i = 0; i < input_size * input_size; ++i) {
        if(i%input_size==0) cout<<"\n";
        cout<<pool_input[i]<<" ";
    }

    cout<<'\n';
       
    // set thread, block dimensions
    const dim3 block_size(TILE_WIDTH, TILE_WIDTH);
    const dim3 num_of_maxpool_blocks(maxpool_output_size/block_size.x+1, maxpool_output_size/block_size.y+1);
    const dim3 num_of_avgpool_blocks(avgpool_output_size/block_size.x+1, avgpool_output_size/block_size.y+1);
    const dim3 num_of_blocks(input_size/block_size.x+1, input_size/block_size.y+1);

    // memory allocation for the device
    float *maxpool_output;
    hipMalloc(&maxpool_output, sizeof(float) * maxpool_output_size * maxpool_output_size);
    
    float *avgpool_output;
    hipMalloc(&avgpool_output, sizeof(float) * avgpool_output_size * avgpool_output_size);
    

    // copy variable to device memory
    hipMemcpy(dev_mem_maxpool_input, pool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_mem_avgpool_input, pool_input, sizeof(float) * input_size * input_size, hipMemcpyHostToDevice);

    // launch CUDA kernels

    // First launch maxpooling kernel
    maxpool<<<num_of_maxpool_blocks, block_size>>>(dev_mem_maxpool_input, maxpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }

    // Then run average pooling
    avgpool<<<num_of_avgpool_blocks, block_size>>>(dev_mem_avgpool_input, avgpool_output, input_size, filter_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if(error!=hipSuccess) {
        fprintf(stderr, "ERROR %s\n", hipGetErrorString(error));
        return 1;
    }
 
    // allocate output buf in main memory
    float *maxpool_output_buf = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    float *avgpool_output_buf = (float*) malloc (sizeof(float)*maxpool_output_size*maxpool_output_size);
    
    // copy results from device to host
    hipMemcpy(maxpool_output_buf, maxpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    hipMemcpy(avgpool_output_buf, avgpool_output, sizeof(float)*maxpool_output_size*maxpool_output_size, hipMemcpyDeviceToHost);
    
    // prints the results
    cout<<"\n========== MAXPOOL OUTPUT ==========\n";
    for (int i = 0; i < maxpool_output_size * maxpool_output_size; ++i) {
        if(i%maxpool_output_size==0) cout<<"\n";
        cout<<maxpool_output_buf[i]<<" ";
    }
    cout<<'\n';

    cout<<"\n========== AVGPOOL OUTPUT ==========\n";
    for (int i = 0; i < avgpool_output_size * avgpool_output_size; ++i) {
        if(i%avgpool_output_size==0) cout<<"\n";
        cout<<avgpool_output_buf[i]<<" ";
    }
    cout<<'\n';

    hipFree(dev_mem_input);
    hipFree(maxpool_output);
    hipFree(avgpool_output);
    free(maxpool_output_buf);
    free(avgpool_output_buf);
    return 0;
}
